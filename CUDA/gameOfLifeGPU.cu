#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../timers/timer.h"

//#include <hip/hip_runtime_api.h>
#define NOLOOPS 100

__global__ void updatedValueGPU(int* prevBlock, int* currentBlock, int nRows, int nColumns)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	int xCoord;
	int yCoord;
	int lowX, lowY, maxX, maxY;
	int activeNeigh;

	if(i< nRows*nColumns)
	{
        xCoord = i/nColumns;
        yCoord = i-xCoord*nColumns;

        lowX = nColumns*((xCoord-1+nRows)%nRows); //the up element on x axis, when -1 goes to nRows-1
        lowY = (yCoord-1 + nColumns)%nColumns; //the up element on y axis, when -1 goes to nColumns-1

        maxX = nColumns*((xCoord+1)%nRows); //the down element on x axis,  when nRows goes to 0
        maxY = (yCoord+1)%nColumns; //the down element on y axis

        activeNeigh =  prevBlock[lowX+lowY]+ prevBlock[lowX+yCoord]+prevBlock[lowX+maxY] //the up 3 neighbors
        +prevBlock[xCoord*nColumns+lowY]+prevBlock[xCoord*nColumns+ maxY] //the middle neighbors
        +prevBlock[maxX+lowY]+ prevBlock[maxX+yCoord]+prevBlock[maxX+maxY]; //the down 3 neighbors


		if(activeNeigh==3 || (activeNeigh==2 && prevBlock[xCoord*nColumns+yCoord] == 1)) //find new value of the cell
		{
			currentBlock[xCoord*nColumns+yCoord]= 1;
		}
		else
		{
			currentBlock[xCoord*nColumns+yCoord] = 0;
		}
	}

}

extern "C" double gameOfLifeGPU(int** h_prevBlock, int** h_currentBlock, int nRows, int nColumns)
{
    int* d_prevBlock;
    int* d_currentBlock;
    int* tempBlock;

	double startTime, finishTime, elapsedTime;
	int memSize;
	const int BLOCK_SIZE = 256;
	int N = nRows*nColumns;
	int k;

    memSize = nRows*nColumns*sizeof(int);

    hipMalloc((void**)&d_prevBlock, memSize); //allocate GPU memory
	hipMalloc((void**)&d_currentBlock, memSize);

	hipMemcpy(d_prevBlock, &h_prevBlock[0][0], memSize, hipMemcpyHostToDevice); //copy CPU memory to GPU

	GET_TIME(startTime);//start time
	
    for(k =0; k <NOLOOPS; k++) //calculate the grid for NOLOOPS
    {
		updatedValueGPU<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_prevBlock, d_currentBlock, nRows, nColumns);

		if(hipGetLastError() !=hipSuccess)
		{
			printf("Error in kernel code\n");
			exit(EXIT_FAILURE);
		}

        tempBlock = d_prevBlock;
        d_prevBlock = d_currentBlock;
        d_currentBlock = tempBlock;
    }
	hipDeviceSynchronize();
	
	GET_TIME(finishTime);
	elapsedTime = finishTime-startTime;

	hipMemcpy(&h_currentBlock[0][0], d_prevBlock, memSize, hipMemcpyDeviceToHost); //copy the results to the CPU memory

    hipFree(d_prevBlock); //free memory
	hipFree(d_currentBlock);

	return elapsedTime;

}
